#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdlib>
#include <ctime>

using namespace std;

__global__ void vecadd(int n, float *x, float *y, float *z)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
  	if (i < n) z[i] = x[i] + y[i];
}

int main(void){
	srand((unsigned int)time(NULL));
	int N=100;
	
	float *x, *y, *z, *d_x, *d_y, *d_z;
	x = new float[N];
	y = new float[N];
	z = new float[N];
	for (int i = 0; i < N; i++){
		x[i] = (float)rand() / RAND_MAX;
		y[i] = (float)rand() / RAND_MAX;
	}

	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));
	hipMalloc(&d_z, N*sizeof(float));
	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);
	
	vecadd <<<(N + 1023) / 1024, 1024>>> (N, d_x, d_y, d_z);
	
	hipMemcpy(z, d_z, N*sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
		cout << x[i] << " + " << y[i] << " = " << z[i] << "\t(" << x[i] + y[i] << ")" << endl;

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	delete[] x;
	delete[] y;
	delete[] z;
	return 0;
}